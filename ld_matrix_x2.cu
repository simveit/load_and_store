
#include <hip/hip_runtime.h>
#include <cstdint>
#include <iostream>

__device__ __forceinline__ void ldmatrix_sync_aligned_m8n8_x2_b16(
    uint32_t &d0, uint32_t &d1, const uint32_t &address) {
  asm volatile("ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0, %1}, [%2];"
               : "=r"(d0), "=r"(d1)
               : "r"(address));
}

__global__ void ldmatrix(uint16_t *value) {
  constexpr int N = 64;
  __shared__ uint16_t smem[2 * N];
  auto tid = threadIdx.x;

  const uint32_t offset_rows = sizeof(uint16_t) * (tid % 8) * 8;
  const uint32_t offset_matrix = sizeof(uint16_t) * ((tid / 8) % 2) * 64;
  const uint32_t offset = offset_rows + offset_matrix;
  const uint32_t address = __cvta_generic_to_shared(smem) + offset;

  for (uint32_t i = tid; i < N; i += blockDim.x) {
    smem[i] = i;
    smem[i + 64] = i + 64;
  }
  __syncthreads();

  uint32_t frag1;
  uint32_t frag2;
  ldmatrix_sync_aligned_m8n8_x2_b16(frag1, frag2, address);

  __syncthreads();

  uint16_t number1 = static_cast<uint16_t>(frag1 & 0xFFFF);
  uint16_t number2 = static_cast<uint16_t>((frag1 >> 16) & 0xFFFF);
  printf("%d -> %d  %d   %d   \n", tid, (int)(smem[2 * tid]), (int)number1,
         (int)number2);
  uint16_t number3 = static_cast<uint16_t>(frag2 & 0xFFFF);
  uint16_t number4 = static_cast<uint16_t>((frag2 >> 16) & 0xFFFF);
  printf("%d -> %d  %d   %d   \n", tid, (int)(smem[2 * tid + 64]), (int)number3,
         (int)number4);
}

int main() {
  uint16_t *d_value;
  hipMalloc(&d_value, sizeof(uint16_t));
  ldmatrix<<<1, 32>>>(d_value);
  hipDeviceSynchronize();
  hipFree(d_value);
  return 0;
}