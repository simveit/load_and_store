
#include <hip/hip_runtime.h>
#include <cstdint>
#include <iostream>

__device__ __forceinline__ void stmatrix_sync_aligned_m8n8_x2_b16(
    uint32_t &d0, uint32_t &d1, const uint32_t &address) {
  asm volatile(
      "stmatrix.sync.aligned.m8n8.x2.shared.b16 [%0], {%1, %2};" ::"r"(address),
      "r"(d0), "r"(d1));
}

__global__ void stmatrix(uint16_t *value) {
  constexpr int N = 64;
  __shared__ uint16_t smem[2 * N];
  auto tid = threadIdx.x;

  const uint32_t offset_rows = sizeof(uint16_t) * (tid % 8) * 8;
  const uint32_t offset_matrix = sizeof(uint16_t) * ((tid / 8) % 2) * 64;
  const uint32_t offset = offset_rows + offset_matrix;
  const uint32_t address = __cvta_generic_to_shared(smem) + offset;

  uint32_t frag1 = 0x00000000;
  frag1 |= (tid * 2 + 0);
  frag1 |= (tid * 2 + 1) << 16;
  uint32_t frag2 = 0x00000000;
  frag2 |= (tid * 2 + 0 + 64);
  frag2 |= (tid * 2 + 1 + 64) << 16;
  __syncthreads();

  stmatrix_sync_aligned_m8n8_x2_b16(frag1, frag2, address);

  __syncthreads();

  uint16_t number1 = static_cast<uint16_t>(frag1 & 0xFFFF);
  uint16_t number2 = static_cast<uint16_t>((frag1 >> 16) & 0xFFFF);
  printf("%d -> %d  %d   %d   \n", tid, (int)(smem[2 * tid]), (int)number1,
         (int)number2);
  uint16_t number3 = static_cast<uint16_t>(frag2 & 0xFFFF);
  uint16_t number4 = static_cast<uint16_t>((frag2 >> 16) & 0xFFFF);
  printf("%d -> %d  %d   %d   \n", tid, (int)(smem[2 * tid + 64]), (int)number3,
         (int)number4);
}

int main() {
  uint16_t *d_value;
  hipMalloc(&d_value, sizeof(uint16_t));
  stmatrix<<<1, 32>>>(d_value);
  hipDeviceSynchronize();
  hipFree(d_value);
  return 0;
}