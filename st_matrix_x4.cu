
#include <hip/hip_runtime.h>
#include <cstdint>
#include <iostream>

__device__ __forceinline__ void stmatrix_sync_aligned_m8n8_x4_b16(
    uint32_t &d0, uint32_t &d1, uint32_t &d2, uint32_t &d3,
    const uint32_t &address) {
  asm volatile(
      "stmatrix.sync.aligned.m8n8.x4.shared.b16 [%0], {%1, %2, %3, %4};" ::"r"(
          address),
      "r"(d0), "r"(d1), "r"(d2), "r"(d3));
}

__global__ void stmatrix(uint16_t *value) {
  constexpr int N = 64;
  __shared__ uint16_t smem[4 * N];
  auto tid = threadIdx.x;

  const uint32_t offset_rows = sizeof(uint16_t) * (tid % 8) * 8;
  const uint32_t offset_matrix = sizeof(uint16_t) * ((tid / 8) % 4) * 64;
  const uint32_t offset = offset_rows + offset_matrix;
  const uint32_t address = __cvta_generic_to_shared(smem) + offset;

  uint32_t frag1 = 0x00000000;
  frag1 |= (tid * 2 + 0);
  frag1 |= (tid * 2 + 1) << 16;
  uint32_t frag2 = 0x00000000;
  frag2 |= (tid * 2 + 0 + 64);
  frag2 |= (tid * 2 + 1 + 64) << 16;
  uint32_t frag3 = 0x00000000;
  frag3 |= (tid * 2 + 0 + 128);
  frag3 |= (tid * 2 + 1 + 128) << 16;
  uint32_t frag4 = 0x00000000;
  frag4 |= (tid * 2 + 0 + 192);
  frag4 |= (tid * 2 + 1 + 192) << 16;
  __syncthreads();

  stmatrix_sync_aligned_m8n8_x4_b16(frag1, frag2, frag3, frag4, address);

  __syncthreads();

  uint16_t number1 = static_cast<uint16_t>(frag1 & 0xFFFF);
  uint16_t number2 = static_cast<uint16_t>((frag1 >> 16) & 0xFFFF);
  printf("%d -> %d  %d   %d   \n", tid, (int)(smem[2 * tid]), (int)number1,
         (int)number2);
  uint16_t number3 = static_cast<uint16_t>(frag2 & 0xFFFF);
  uint16_t number4 = static_cast<uint16_t>((frag2 >> 16) & 0xFFFF);
  printf("%d -> %d  %d   %d   \n", tid, (int)(smem[2 * tid + 64]), (int)number3,
         (int)number4);
  uint16_t number5 = static_cast<uint16_t>(frag3 & 0xFFFF);
  uint16_t number6 = static_cast<uint16_t>((frag3 >> 16) & 0xFFFF);
  printf("%d -> %d  %d   %d   \n", tid, (int)(smem[2 * tid + 128]),
         (int)number5, (int)number6);
  uint16_t number7 = static_cast<uint16_t>(frag4 & 0xFFFF);
  uint16_t number8 = static_cast<uint16_t>((frag4 >> 16) & 0xFFFF);
  printf("%d -> %d  %d   %d   \n", tid, (int)(smem[2 * tid + 192]),
         (int)number7, (int)number8);
}

int main() {
  uint16_t *d_value;
  hipMalloc(&d_value, sizeof(uint16_t));
  stmatrix<<<1, 32>>>(d_value);
  hipDeviceSynchronize();
  hipFree(d_value);
  return 0;
}